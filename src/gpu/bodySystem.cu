#include "hip/hip_runtime.h"

#include <cmath>

#include <GL/glew.h>
#if defined(__APPLE__) || defined(MACOSX)
#pragma clang diagnostic ignored "-Wdeprecated-declarations"
#include <GLUT/glut.h>
#else
#include <GL/freeglut.h>
#endif

// CUDA standard includes
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <vector_types.hpp>
#include <hip/hip_runtime_api.h>

__constant__ float softeningSquared;
__constant__ double softeningSquared_fp64;

hipError_t setSofteningSquared(float softeningSq)
{
    return hipMemcpyToSymbol(HIP_SYMBOL(softeningSquared),
                              &softeningSq,
                              sizeof(float), 0,
                              hipMemcpyHostToDevice);
}

hipError_t setSofteningSquared(double softeningSq)
{
    return hipMemcpyToSymbol(HIP_SYMBOL(softeningSquared_fp64),
                              &softeningSq,
                              sizeof(double), 0,
                              hipMemcpyHostToDevice);
}

template<class T>
struct SharedMemory
{
    __device__ inline operator       T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};

template<typename T>
__device__ T rsqrt_T(T x)
{
return rsqrt(x);
}

template<>
__device__ float rsqrt_T<float>(float x)
{
    return rsqrtf(x);
}

template<>
__device__ double rsqrt_T<double>(double x)
{
    return rsqrt(x);
}


// Macros to simplify shared memory addressing
#define SX(i) sharedPos[i+blockDim.x*threadIdx.y]
// This macro is only used when multithreadBodies is true (below)
#define SX_SUM(i,j) sharedPos[i+blockDim.x*j]

template <typename T>
__device__ T getSofteningSquared()
{
    return softeningSquared;
}
template <>
__device__ double getSofteningSquared<double>()
{
    return softeningSquared_fp64;
}

template <typename T>
struct DeviceData
{
    T *dPos[2]; // mapped host pointers
    T *dVel;
    hipEvent_t  event;
    unsigned int offset;
    unsigned int numBodies;
};


template <typename T>
__device__ typename vec3<T>::Type
bodyBodyInteraction(typename vec3<T>::Type ai,
                    typename vec4<T>::Type bi,
                    typename vec4<T>::Type bj)
{
    // r_ij  [3 FLOPS]
    typename vec3<T>::Type r = bj - bi;

    // distSqr = dot(r_ij, r_ij) + EPS^2  [6 FLOPS]
    T distSqr = r.dot();
    distSqr += getSofteningSquared<T>();

    // invDistCube =1/distSqr^(3/2)  [4 FLOPS (2 mul, 1 sqrt, 1 inv)]
    T invDist = rsqrt_T(distSqr);
    T invDistCube =  invDist * invDist * invDist;

    // s = m_j * invDistCube [1 FLOP]
    T s = bj.w * invDistCube;

    // a_i =  a_i + s * r_ij [6 FLOPS]
    ai += r * s;

    return ai;
}

template <typename T>
__device__ typename vec3<T>::Type
computeBodyAccel(typename vec4<T>::Type bodyPos,
                 typename vec4<T>::VecType *positions,
                 int numTiles)
{
    typename vec4<T>::VecType *sharedPos = SharedMemory<typename vec4<T>::VecType>();

    typename vec3<T>::Type acc = {0.0f, 0.0f, 0.0f};

    for (int tile = 0; tile < numTiles; tile++)
    {
        sharedPos[threadIdx.x] = positions[tile * blockDim.x + threadIdx.x];

        __syncthreads();

        // This is the "tile_calculation" from the GPUG3 article.
#pragma unroll 128

        for (unsigned int counter = 0; counter < blockDim.x; counter++)
        {
            acc = bodyBodyInteraction<T>(acc, bodyPos, sharedPos[counter]);
        }

        __syncthreads();
    }

    return acc;
}

template<typename T>
__global__ void
integrateBodies(typename vec4<T>::VecType *__restrict__ newPos,
                typename vec4<T>::VecType *__restrict__ oldPos,
                typename vec4<T>::VecType *             vel,
                unsigned int deviceOffset, unsigned int deviceNumBodies,
                T deltaTime, T damping, int numTiles)
{
    for ( int index = blockIdx.x * blockDim.x + threadIdx.x;
              index < deviceNumBodies;
              index += blockDim.x * gridDim.x )
    {
        typename vec4<T>::Type position = oldPos[deviceOffset + index];

        typename vec3<T>::Type accel = computeBodyAccel<T>(position,
                                                           oldPos,
                                                           numTiles);

        // acceleration = force / mass;
        // new velocity = old velocity + acceleration * deltaTime
        // note we factor out the body's mass from the equation, here and in bodyBodyInteraction
        // (because they cancel out).  Thus here force == acceleration
        typename vec4<T>::Type velocity = vel[deviceOffset + index];

        velocity += accel * deltaTime;
        velocity *= damping;

        // new position = old position + velocity * deltaTime
        position += velocity * deltaTime;

        // store new position and velocity
        typename vec4<T>::VecType newPosition = {position.x, position.y, position.z, position.w};
        typename vec4<T>::VecType newVelocity = {velocity.x, velocity.y, velocity.z, velocity.w};
        newPos[deviceOffset + index] = newPosition;
        vel[deviceOffset + index]    = newVelocity;
    }
}

template <typename T>
void integrateNbodySystem(DeviceData<T> *deviceData,
                          hipGraphicsResource **pgres,
                          unsigned int currentRead,
                          float deltaTime,
                          float damping,
                          unsigned int numBodies,
                          unsigned int numDevices,
                          int blockSize,
                          bool bUsePBO)
{
    if (bUsePBO)
    {
        cudaGraphicsResourceSetMapFlags(pgres[currentRead], cudaGraphicsMapFlagsReadOnly);
        cudaGraphicsResourceSetMapFlags(pgres[1-currentRead], cudaGraphicsMapFlagsWriteDiscard);
        hipGraphicsMapResources(2, pgres, 0);
        size_t bytes;
        hipGraphicsResourceGetMappedPointer((void **)&(deviceData[0].dPos[currentRead]), &bytes, pgres[currentRead]);
        hipGraphicsResourceGetMappedPointer((void **)&(deviceData[0].dPos[1-currentRead]), &bytes, pgres[1-currentRead]);
    }

    for (unsigned int dev = 0; dev != numDevices; dev++)
    {
        if (numDevices > 1)
        {
            hipSetDevice(dev);
        }

        int numBlocks = (deviceData[dev].numBodies + blockSize-1) / blockSize;
        int numTiles = (numBodies + blockSize - 1) / blockSize;
        int sharedMemSize = blockSize * 4 * sizeof(T); // 4 floats for pos

        integrateBodies<T><<< numBlocks, blockSize, sharedMemSize >>>
                        ((typename vec4<T>::VecType *)deviceData[dev].dPos[1-currentRead],
                         (typename vec4<T>::VecType *)deviceData[dev].dPos[currentRead],
                         (typename vec4<T>::VecType *)deviceData[dev].dVel,
                                                   deviceData[dev].offset,
                                                   deviceData[dev].numBodies,
                                                   deltaTime, damping, numTiles);

        if (numDevices > 1)
        {
            hipEventRecord(deviceData[dev].event);
            // MJH: Hack on older driver versions to force kernel launches to flush!
            hipStreamQuery(0);
        }

        // check if kernel invocation generated an error
        //getLastCudaError("Kernel execution failed");
    }

    if (numDevices > 1)
    {
        for (unsigned int dev = 0; dev < numDevices; dev++)
        {
            hipEventSynchronize(deviceData[dev].event);
        }
    }

    if (bUsePBO)
    {
        hipGraphicsUnmapResources(2, pgres, 0);
    }
}


// Explicit specializations needed to generate code
template void integrateNbodySystem<float>(DeviceData<float> *deviceData,
                                          hipGraphicsResource **pgres,
                                          unsigned int currentRead,
                                          float deltaTime,
                                          float damping,
                                          unsigned int numBodies,
                                          unsigned int numDevices,
                                          int blockSize,
                                          bool bUsePBO);

template void integrateNbodySystem<double>(DeviceData<double> *deviceData,
                                           hipGraphicsResource **pgres,
                                           unsigned int currentRead,
                                           float deltaTime,
                                           float damping,
                                           unsigned int numBodies,
                                           unsigned int numDevices,
                                           int blockSize,
                                           bool bUsePBO);
